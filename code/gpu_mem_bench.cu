
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <unistd.h>



//define the check err
#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
	    hipGetErrorString(err),__FILE__,__LINE__);	\
    exit(-1);						\
  }                                                     \

//define the maximum operations for the loop 
unsigned long MAX_OPERATIONS = 40000000;

//size for 1MB
const long MEGABYTE = 1048576;

__global__ void gpu_iops(unsigned long max_ops) {


    
}

//get the parameter
int main(int argc, char *argv[]) {
  
    char c;  
    char test = 'B';
    char rw = 'R';
    while ( (c = getopt(argc, argv, "r:t:") ) != -1)
    {
        switch (c) 
        {
            case 'r':
                rw = optarg[0];
                break;
            case 't':
                test = optarg[0];
                break;
            default:
                printf("nothing\n");
                return -1;
        }
    }
//define the time variables
    struct timeval tv;
    long long startTime, stopTime;
    double seconds;
	
//define the cuda err
    hipError_t err;
 
//define the memory pointer
 
    unsigned char *d_mem_pointer;
    unsigned char *mem_pointer;
	
    hipMemcpyKind dir = hipMemcpyHostToDevice;
	
	//direction for read and write
    if(rw == 'R')
    {
      dir = hipMemcpyDeviceToHost;
    }
	
    else if(rw == 'W')
    {
      dir - hipMemcpyHostToDevice;
    }
    
    //test the 1B size block
    if(test == 'B')
    {
      
      err = hipMalloc((void **) &d_mem_pointer, sizeof(unsigned char)*MEGABYTE);
      CHECK_ERR(err);
      mem_pointer = (unsigned char *)malloc(sizeof(unsigned char)*1);
      gettimeofday(&tv, NULL);
      startTime = tv.tv_sec*1000000LL + tv.tv_usec;
    
      for(unsigned long i = 0; i<MEGABYTE; i++)
      {
//write operation
        if(rw == 'W')
          err = hipMemcpy((void *)&d_mem_pointer[i], (void *)mem_pointer, 1, dir);
//read operation  
      else if(rw == 'R')
          err = hipMemcpy((void *)mem_pointer, (void *)&d_mem_pointer[i], 1, dir);
        CHECK_ERR(err);
      }
      
      gettimeofday(&tv, NULL);
//record the stop time 
      stopTime = tv.tv_sec*1000000LL + tv.tv_usec;
	  
//totally time cost 
 
      seconds = (stopTime-startTime)/1000000.0;
      printf("Operation: %c\tMessage Size:1%c\tBandwidth: %lf MB/S \n", rw, test, 1.0/(seconds)); 
    }
//test the 1K block size
    else if(test == 'K')
    {
      err = hipMalloc((void **) &d_mem_pointer, sizeof(unsigned char)*256*MEGABYTE);
      CHECK_ERR(err);
      mem_pointer = (unsigned char *)malloc(sizeof(unsigned char)*1024);
      gettimeofday(&tv, NULL);
      startTime = tv.tv_sec*1000000LL + tv.tv_usec;
    
      for(unsigned long i = 0; i<256*MEGABYTE/1024; i++)
      {
// test the write operation 
        if(rw == 'W')
          err = hipMemcpy((void *)&d_mem_pointer[i*1024], (void *)mem_pointer, 1024, dir);
//test the read operation 
        else if(rw == 'R')
          err = hipMemcpy((void *)mem_pointer, (void *)&d_mem_pointer[i*1024], 1024, dir);
        CHECK_ERR(err);
      }
      
      gettimeofday(&tv, NULL);
//record the stop time
      stopTime = tv.tv_sec*1000000LL + tv.tv_usec;
// totally time cost 
      seconds = (stopTime-startTime)/1000000.0;
      printf("Operation:%c\tMessage Size:1%cB\tBandwidth: %lf MB/S \n", rw, test, (256.0/1024.0)/(seconds)); 
    }
//test 1MB size Block
    else if(test == 'M')
    {
      err = hipMalloc((void **) &d_mem_pointer, sizeof(unsigned char)*512*MEGABYTE);
      CHECK_ERR(err);
      mem_pointer = (unsigned char *)malloc(sizeof(unsigned char)*MEGABYTE);
      gettimeofday(&tv, NULL);
      startTime = tv.tv_sec*1000000LL + tv.tv_usec;
    
      for(unsigned long i = 0; i<512*10; i++)
      {
//write operation
        if(rw == 'W')
          err = hipMemcpy((void *)&d_mem_pointer[(i*MEGABYTE)%(512*MEGABYTE)], (void *)mem_pointer, MEGABYTE, dir);
//read operation 
        else if(rw == 'R')
          err = hipMemcpy((void *)mem_pointer, (void *)&d_mem_pointer[(i*MEGABYTE)%(512*MEGABYTE)], MEGABYTE, dir);
        CHECK_ERR(err);
      }
      
      gettimeofday(&tv, NULL);
// record the stop time
      stopTime = tv.tv_sec*1000000LL + tv.tv_usec;
//ttoally time cost 
      seconds = (stopTime-startTime)/1000000.0;
      printf("Operation:%c\tMessage Size:1%cB \tBandwidth:%lf MB/S\n", rw, test, (512*10)/(seconds)); 
    }
    err = hipFree(d_mem_pointer);
    CHECK_ERR(err);
}
