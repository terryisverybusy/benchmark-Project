#include "hip/hip_runtime.h"

#include <sys/time.h>
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>


                                                     

// this is the max iterations decide to do in the loop
unsigned long MAX_OPS = 20000000;

__global__ void gpu_iops(unsigned long max_ops) {


    
    int ab=1;
    int bb=1;
    int cb=1;
    int db=1;
    int eb=1;
    int fb=1;
    int gb=1;
    int hb=1;
    int ib=1;
    int jb=1;
    int kb=1;
    int lb=1;
    int mb=1;
    int nb=1;
    int ob=1;
    int pb=1;
    int qb=1;
    int rb=1;
    int sb=1;
    int tb=1;
    int ub=1;
    int vb=1;
    int wb=1;
    int xb=1;
   // 24 interger calculation 
    for(unsigned long count=0; count<max_ops; count++)
    {
        ab=ab+12;
        bb=bb+22;
        cb=cb+32;
        db=db+42;
        eb=eb+52;
        fb=fb+62;
        gb=gb+72;
        hb=hb+82;
        ib=ib+92;
        jb=jb+10;
        kb=kb+11;
        lb=lb+12;
        mb=mb*13;
        nb=nb*14;
        ob=ob*15;
        pb=pb*16;
        qb=qb*17;
        rb=rb*18;
        sb=sb*19;
        tb=tb*20;
        ub=ub*21;
        vb=vb*22;
        wb=wb*23;
        xb=xb*24;
    }
}

__global__ void gpu_flops(unsigned long max_ops) {


    float ae=0.2;
    float be=0.2;
    float ce=0.2;
    float de=0.2;
    float ee=0.2;
    float fe=0.2;
    float ge=0.2;
    float he=0.2;
    float ie=0.2;
    float je=0.2;
    float ke=0.2;
    float le=0.2;
    float me=0.2;
    float ne=0.2;
    float oe=0.2;
    float pe=0.2;
    float qe=0.2;
    float re=0.2;
    float se=0.2;
    float te=0.2;
    float ue=0.2;
    float ve=0.2;
    float we=0.2;
    float xe=0.2;

//24  floating point calculation
    for(unsigned long count=0; count<max_ops; count++)
    {
        ae=ae+1.11;
        be=be+2.22;
        ce=ce+3.33;
        de=de+4.44;
        ee=ee+5.52;
        fe=fe+6.61;
        ge=ge+7.72;
        he=he+8.83;
        ie=ie+9.94;
        je=je+10.10;
        ke=ke+11.11;
        le=le+12.12;
        me=me*13.13;
        ne=ne*14.14;
        oe=oe*15.15;
        pe=pe*16.16;
        qe=qe*17.17;
        re=re*18.18;
        se=se*19.19;
        te=te*20.20;
        ue=ue*21.21;
        ve=ve*22.22;
        we=we*23.23;
        xe=xe*24.24;
    }
}

int main(int argc, char *argv[]) {
//decide the character  
    char c; 

//decide the maximum thread 
    int threads = 1024;
    char test = 'I';

//get the parameter
    while ( (c = getopt(argc, argv, "n:l:t:") ) != -1) 
    {
        switch (c) 
        {
            case 'n':
                threads = atoi(optarg);
                break;
            case 'l':
                MAX_OPS = atol(optarg);
                break;
            case 't':
                test = optarg[0];
                break;
            default:
                printf("Usage\n");
                return -1;
        }
    }
   
//set the time parameter
    struct timeval tv;
    long long start, stop;
    double secs;

    if(test == 'I')
    {
      gettimeofday(&tv, NULL);
//get the start time
      start = tv.tv_sec*1000000LL + tv.tv_usec;   
      gpu_iops<<< ceil(threads/1024), 1024 >>>(MAX_OPS);        
      hipDeviceSynchronize(); 
      gettimeofday(&tv, NULL);
//get the stop time
      stop = tv.tv_sec*1000000LL + tv.tv_usec;
//get the totally time cost 
      secs = (stop-start)/1000000.0;
      printf("Time taken: %lf\n", secs);
      printf("IOPS\t%lf\n", (MAX_OPS*24.*threads)/(secs*10000000000.)); 
    }
    else if(test == 'F')
    {
      gettimeofday(&tv, NULL);
//get the start time
      start = tv.tv_sec*1000000LL + tv.tv_usec;
      
      gpu_flops<<< ceil(threads/1024), 1024 >>>(MAX_OPS);        
      hipDeviceSynchronize();
// get the stop time 
      gettimeofday(&tv, NULL);
      stop = tv.tv_sec*1000000LL + tv.tv_usec;
//get the totally time cost 
      secs = (stop-start)/1000000.0;
      printf("Time taken: %lf\n", secs);
      printf("GFLOPS\t%lf\n", (MAX_OPS*24.*threads)/(secs*10000000000.)); 
    }
}
